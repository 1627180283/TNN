#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "tnn/device/cuda/utils/cuda_blob_converter_kernel.cuh"

namespace TNN_NS {

#define ELEMENT_PER_THREAD 4
#define THREAD_PER_BLOCK 128

inline __device__ unsigned char fp32_to_u8_sat(float in) {
    int x = __float2int_rn(in);
    x = x > 255 ? 255 : x;
    x = x > 0 ? x : 0;
    return (unsigned char)(x);
}

__global__ void blob_to_bgr_kernel(int CHW, int HW, const float* __restrict__ src, unsigned char *dst,
        int channels, float *scale, float *bias) {
    const int offset = ELEMENT_PER_THREAD * THREAD_PER_BLOCK * blockIdx.x + threadIdx.x;

    src += offset + blockIdx.y * CHW;
    dst += offset * channels + blockIdx.y * CHW;
    int channels_coef = channels - 1;

    #pragma unroll
    for (int c = 0; c < channels; ++c) {
        unsigned char data_ld[ELEMENT_PER_THREAD];
        #pragma unroll
        for (int i = 0; i < ELEMENT_PER_THREAD; ++i) {
            if (i * THREAD_PER_BLOCK + offset < HW) {
                data_ld[i] = fp32_to_u8_sat(src[i * THREAD_PER_BLOCK + (channels_coef - c) * HW]
                                                * scale[channels_coef - c]
                                                + bias[channels_coef - c]);
            }
        }
        #pragma unroll
        for (int i = 0; i < ELEMENT_PER_THREAD; ++i) {
            if (i * THREAD_PER_BLOCK + offset < HW) {
                dst[c + i * THREAD_PER_BLOCK * channels] = data_ld[i];
            }
        }
    }
}

__global__ void blob_to_gray_kernel(int count, const float *src, unsigned char *dst, float scale, float bias) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < count)
        dst[index] = fp32_to_u8_sat(scale * src[index] + bias);
}

__global__ void bgr_to_blob_kernel(int CHW, int HW, const unsigned char* __restrict__ src, float *dst,
        int channels, float *scale, float *bias) {
    const int offset = ELEMENT_PER_THREAD * THREAD_PER_BLOCK * blockIdx.x + threadIdx.x;

    src += offset * channels + blockIdx.y * CHW;
    dst += offset + blockIdx.y * CHW;

    #pragma unroll
    for (int c = 0; c < channels; ++c) {
        float data_ld[ELEMENT_PER_THREAD];
        #pragma unroll
        for (int i = 0; i < ELEMENT_PER_THREAD; ++i) {
            if (i * THREAD_PER_BLOCK + offset < HW) {
                data_ld[i] = (src[i * THREAD_PER_BLOCK * channels + c] * scale[c] + bias[c]);
            }
        }
        #pragma unroll
        for (int i = 0; i < ELEMENT_PER_THREAD; ++i) {
            if (i * THREAD_PER_BLOCK + offset < HW) {
                dst[c * HW + i * THREAD_PER_BLOCK] = data_ld[i];
            }
        }
    }
}

__global__ void gray_to_blob_kernel(int count, const unsigned char *src, float *dst, float scale, float bias) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < count)
        dst[index] = scale * src[index] + bias;
}

void BlobToBGR(int batch, int CHW, int HW, const float *src, unsigned char *dst, hipStream_t stream,
        int channels, float *scale, float *bias) {
    dim3 grid;
    grid.x = (HW + ELEMENT_PER_THREAD * THREAD_PER_BLOCK - 1) / (ELEMENT_PER_THREAD * THREAD_PER_BLOCK);
    grid.y = batch;
    blob_to_bgr_kernel<<<grid, THREAD_PER_BLOCK, 0, stream>>>(
        CHW, HW, src, dst, channels, scale, bias);
}

void BlobToGray(int count, const float *src, unsigned char *dst, hipStream_t stream, float scale, float bias) {
    const int BLOCK_NUM = (count + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
    blob_to_gray_kernel<<<BLOCK_NUM, THREAD_PER_BLOCK, 0, stream>>>(count, src, dst, scale, bias);
}

void BGRToBlob(int batch, int CHW, int HW, const unsigned char *src, float *dst, hipStream_t stream,
        int channels, float *scale, float* bias) {
    dim3 grid;
    grid.x = (HW + ELEMENT_PER_THREAD * THREAD_PER_BLOCK - 1) / (ELEMENT_PER_THREAD * THREAD_PER_BLOCK);
    grid.y = batch;
    bgr_to_blob_kernel<<<grid, THREAD_PER_BLOCK, 0, stream>>>(
        CHW, HW, src, dst, channels, scale, bias);
}

void GrayToBlob(int count, const unsigned char *src, float *dst, hipStream_t stream, float scale, float bias) {
    const int BLOCK_NUM = (count + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
    gray_to_blob_kernel<<<BLOCK_NUM, THREAD_PER_BLOCK, 0, stream>>>(count, src, dst, scale, bias);
}

}  //  namespace TNN_NS
